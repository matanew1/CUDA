#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__global__ void computeHistogram(int* array, int* hist, int* size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < *size) {
      hist[array[tid]]++;
    }
}

int computeOnGPU(int *local_array, int* split_size, int* hist) {

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate data on device
    int* d_array = NULL;
    int* d_hist = NULL;

    err = hipMalloc((void**)&d_array, (*split_size) * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void**)&d_hist, RANGE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Copy data to the device
    err = hipMemcpy(d_array, local_array, (*split_size), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    //Copy data to the device
    err = hipMemcpy(d_hist, hist, RANGE, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Initialize hist on device
    computeHistogram<<<NUM_BLOCKS, NUM_THREADS>>>(d_array, d_hist, split_size);
    
    hipMemcpy(hist, d_hist, RANGE * sizeof(int), hipMemcpyDeviceToHost);
    // for(int i=0; i < RANGE; i++) 
    // {
    //   printf("\n%d %d", i,hist[i]);
    // }

    // Free device global memory
    hipFree(d_array);
    hipFree(d_hist);

    printf("Done\n");
    return 0;
}

