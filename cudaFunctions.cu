#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

  __global__  void buildHist(int *h, int *temp) {
    int index = threadIdx.x;
  
    for (int i = 0; i < NUM_THREADS;   i++)
      h[index] += temp[index + i*RANGE];
  }
  __global__  void buildTemp(int *A, int *temp) {
  
    int index = threadIdx.x;
    int offset_A = SIZE / NUM_THREADS * index;
    int offset_temp = RANGE * index;
  
    for (int i = 0;    i < SIZE / NUM_THREADS;   i++) {
      int value = A[offset_A + i];
      temp[offset_temp + value]++;
    }
  
  }
  __global__  void initHist(int * h) {
    int index = threadIdx.x;
    printf("[%d]",index);
    h[index] = 0;
  
  }
  __global__  void initTemp(int * temp) {
  
    int index = threadIdx.x;
    int offset = RANGE * index;
    for (int i = 0;    i < RANGE;   i++)
      temp[offset + i] = 0;
  }

int computeOnGPU(int *data, int numElements, int* hist) {

    dim3 gridSize(10); // 10 blocks in the grid
    dim3 blockSize(20);  // 20 threads per block

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t size = numElements * sizeof(int);

    // Allocate data on device
    int* device_data = NULL;
    err = hipMalloc((void **)&device_data, SIZE / 2 * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Allocate hist on device
    int* device_hist = NULL;
    err = hipMalloc((void **)&device_hist, RANGE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Copy data to the device
    err = hipMemcpy(device_data, data, SIZE / 2, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    //Copy data to the device
    err = hipMemcpy(device_hist, hist, RANGE, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Initialize hist on device
    initHist<<<gridSize, blockSize>>>(device_hist);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Initialize data on device
    initTemp <<< gridSize, blockSize >>> (device_data);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Unify the results
    buildHist<<< gridSize, blockSize >>>(device_hist, device_data);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Copy the final histogram to the host
    err = hipMemcpy(hist, device_hist, RANGE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }


    // Free device global memory
    err = hipFree(device_data);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    err = hipFree(device_hist);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

