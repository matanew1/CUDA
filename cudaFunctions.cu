#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__global__ void computeHistogram(int* array, int* hist, int* size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    for (int i = 0; i < NUM_THREADS; i++)
      h[tid] += temp[tid + i*RANGE];
}

int computeOnGPU(int *local_array, int* split_size, int* hist) {

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate data on device
    int* d_array = NULL;
    int* d_hist = NULL;

    err = hipMalloc((void**)&d_array, (*split_size) * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void**)&d_hist, RANGE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Copy data to the device
    err = hipMemcpy(d_array, local_array, (*split_size) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Initialize hist on device
    err = hipMemset(d_hist, 0, RANGE * sizeof(int));
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Launch kernel
    computeHistogram<<<10, 20>>>(d_array, d_hist, split_size);

    // Copy histogram data back to host
    err = hipMemcpy(hist, d_hist, RANGE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free device global memory
    hipFree(d_array);
    hipFree(d_hist);

    printf("Done\n");
    return 0;
}
