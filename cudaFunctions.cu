#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"


/*
  tid - is the thread index within the grid.
  blockIdx.x - is the index of the current block in the x dimension.
  blockDim.x - is the size of each block in the x dimension.
  threadIdx.x - is the thread index within the block.
  stride - is the total number of threads in the grid.
*/

__global__  void initArr(int * h) {

  int index = threadIdx.x;
  h[index] = 0;

}


__global__ void histogram_kernel(int *input, int *histogram, int split_size)
{
  // gridDim.x = 10
  // blockDim.x = 20

    int tid = blockIdx.x * blockDim.x + threadIdx.x; // range 0 - 199
    int stride = blockDim.x * gridDim.x; // 200

    /*        tid = 0
      0 -> 200 -> 400 ->...->499800
    */
    while (tid < split_size) 
    {
      histogram[input[tid]]++;
      tid += stride;
    }
}

int computeOnGPU(int *local_array, int *split_size, int *hist)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate data on device
    int *d_array = NULL;
    int *d_hist = NULL;

    // Malloc device local_array and hist on device
    err = hipMalloc((void **)&d_array, (*split_size) * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_hist, RANGE * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data to the device
    err = hipMemcpy(d_array, local_array, (*split_size) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Initialize vectors on device
    initArr <<< 1 , RANGE >>> (d_hist);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Initialize hist on device
    histogram_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(d_array, d_hist, *split_size);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the result back to the host
    err = hipMemcpy(hist, d_hist, RANGE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_array);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_hist);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}
