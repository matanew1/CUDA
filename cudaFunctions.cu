#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__global__ void calculateHistogram(int* data, int* histogram, int* size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < *size) {
        atomicAdd(&histogram[data[tid]], 1);
    }
}

int computeOnGPU(int *data, int* split_size, int* histogram) {

    int* dev_data;
    int* dev_histogram;

        // Allocate memory on GPU
    hipMalloc((void**)&dev_data, (*split_size) * sizeof(int));
    hipMalloc((void**)&dev_histogram, NUM_BINS * sizeof(int));

    // Copy data from host to device
    hipMemcpy(dev_data, data, (*split_size) * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(dev_histogram, 0, NUM_BINS * sizeof(int));

    // Launch CUDA kernels
    int num_blocks = (*split_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    calculateHistogram<<<num_blocks, BLOCK_SIZE>>>(dev_data, dev_histogram, split_size);
    
    // Copy histogram from device to host
    hipMemcpy(histogram, dev_histogram, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);

    // Free device global memory
    hipFree(dev_data);
    hipFree(dev_histogram);

    printf("Done\n");
    return 0;
}
